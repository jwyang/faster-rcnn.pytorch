#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include <stdio.h>
#include <math.h>
#include <float.h>
#include "roi_align_kernel.h"

#define CUDA_1D_KERNEL_LOOP(i, n)                            \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
            i += blockDim.x * gridDim.x)


    __global__ void ROIAlignForward(const int nthreads, const float* bottom_data, const float spatial_scale, const int height, const int width,
                                    const int channels, const int aligned_height, const int aligned_width, const float* bottom_rois, float* top_data) {
        CUDA_1D_KERNEL_LOOP(index, nthreads) {
            // (n, c, ph, pw) is an element in the aligned output
            // int n = index;
            // int pw = n % aligned_width;
            // n /= aligned_width;
            // int ph = n % aligned_height;
            // n /= aligned_height;
            // int c = n % channels;
            // n /= channels;

            int pw = index % aligned_width;
            int ph = (index / aligned_width) % aligned_height;
            int c  = (index / aligned_width / aligned_height) % channels;
            int n  = index / aligned_width / aligned_height / channels;

            // bottom_rois += n * 5;
            float roi_batch_ind = bottom_rois[n * 5 + 0];
            float roi_start_w = bottom_rois[n * 5 + 1] * spatial_scale;
            float roi_start_h = bottom_rois[n * 5 + 2] * spatial_scale;
            float roi_end_w = bottom_rois[n * 5 + 3] * spatial_scale;
            float roi_end_h = bottom_rois[n * 5 + 4] * spatial_scale;

            // Force malformed ROIs to be 1x1
            float roi_width = fmaxf(roi_end_w - roi_start_w + 1., 0.);
            float roi_height = fmaxf(roi_end_h - roi_start_h + 1., 0.);
            float bin_size_h = roi_height / (aligned_height - 1.);
            float bin_size_w = roi_width / (aligned_width - 1.);

            float h = (float)(ph) * bin_size_h + roi_start_h;
            float w = (float)(pw) * bin_size_w + roi_start_w;

            int hstart = fminf(floor(h), height - 2);
            int wstart = fminf(floor(w), width - 2);

            int img_start = roi_batch_ind * channels * height * width;

            // bilinear interpolation
            if (h < 0 || h >= height || w < 0 || w >= width) {
                top_data[index] = 0.;
            } else {
                float h_ratio = h - (float)(hstart);
                float w_ratio = w - (float)(wstart);
                int upleft = img_start + (c * height + hstart) * width + wstart;
                int upright = upleft + 1;
                int downleft = upleft + width;
                int downright = downleft + 1;

                top_data[index] = bottom_data[upleft] * (1. - h_ratio) * (1. - w_ratio)
                    + bottom_data[upright] * (1. - h_ratio) * w_ratio
                    + bottom_data[downleft] * h_ratio * (1. - w_ratio)
                    + bottom_data[downright] * h_ratio * w_ratio;
            }
        }
    }


    int ROIAlignForwardLaucher(const float* bottom_data, const float spatial_scale, const int num_rois, const int height, const int width,
                               const int channels, const int aligned_height, const int aligned_width, const float* bottom_rois, float* top_data, hipStream_t stream) {
        const int kThreadsPerBlock = 1024;
        const int output_size = num_rois * aligned_height * aligned_width * channels;
        hipError_t err;


        ROIAlignForward<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream>>>(
          output_size, bottom_data, spatial_scale, height, width, channels,
          aligned_height, aligned_width, bottom_rois, top_data);

        err = hipGetLastError();
        if(hipSuccess != err) {
            fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
            exit( -1 );
        }

        return 1;
    }


    __global__ void ROIAlignBackward(const int nthreads, const float* top_diff, const float spatial_scale, const int height, const int width,
                                     const int channels, const int aligned_height, const int aligned_width, float* bottom_diff, const float* bottom_rois) {
        CUDA_1D_KERNEL_LOOP(index, nthreads) {

            // (n, c, ph, pw) is an element in the aligned output
            int pw = index % aligned_width;
            int ph = (index / aligned_width) % aligned_height;
            int c  = (index / aligned_width / aligned_height) % channels;
            int n  = index / aligned_width / aligned_height / channels;

            float roi_batch_ind = bottom_rois[n * 5 + 0];
            float roi_start_w = bottom_rois[n * 5 + 1] * spatial_scale;
            float roi_start_h = bottom_rois[n * 5 + 2] * spatial_scale;
            float roi_end_w = bottom_rois[n * 5 + 3] * spatial_scale;
            float roi_end_h = bottom_rois[n * 5 + 4] * spatial_scale;
            /* int roi_start_w = round(bottom_rois[1] * spatial_scale); */
            /* int roi_start_h = round(bottom_rois[2] * spatial_scale); */
            /* int roi_end_w = round(bottom_rois[3] * spatial_scale); */
            /* int roi_end_h = round(bottom_rois[4] * spatial_scale); */

            // Force malformed ROIs to be 1x1
            float roi_width = fmaxf(roi_end_w - roi_start_w + 1., 0.);
            float roi_height = fmaxf(roi_end_h - roi_start_h + 1., 0.);
            float bin_size_h = roi_height / (aligned_height - 1.);
            float bin_size_w = roi_width / (aligned_width - 1.);

            float h = (float)(ph) * bin_size_h + roi_start_h;
            float w = (float)(pw) * bin_size_w + roi_start_w;

            int hstart = fminf(floor(h), height - 2);
            int wstart = fminf(floor(w), width - 2);

            int img_start = roi_batch_ind * channels * height * width;

            // bilinear interpolation
            if (!(h < 0 || h >= height || w < 0 || w >= width)) {
                float h_ratio = h - (float)(hstart);
                float w_ratio = w - (float)(wstart);
                int upleft = img_start + (c * height + hstart) * width + wstart;
                int upright = upleft + 1;
                int downleft = upleft + width;
                int downright = downleft + 1;

                atomicAdd(bottom_diff + upleft, top_diff[index] * (1. - h_ratio) * (1 - w_ratio));
                atomicAdd(bottom_diff + upright, top_diff[index] * (1. - h_ratio) * w_ratio);
                atomicAdd(bottom_diff + downleft, top_diff[index] * h_ratio * (1 - w_ratio));
                atomicAdd(bottom_diff + downright, top_diff[index] * h_ratio * w_ratio);
            }
        }
    }

    int ROIAlignBackwardLaucher(const float* top_diff, const float spatial_scale, const int batch_size, const int num_rois, const int height, const int width,
                                const int channels, const int aligned_height, const int aligned_width, const float* bottom_rois, float* bottom_diff, hipStream_t stream) {
        const int kThreadsPerBlock = 1024;
        const int output_size = num_rois * aligned_height * aligned_width * channels;
        hipError_t err;

        ROIAlignBackward<<<(output_size + kThreadsPerBlock - 1) / kThreadsPerBlock, kThreadsPerBlock, 0, stream>>>(
          output_size, top_diff, spatial_scale, height, width, channels,
          aligned_height, aligned_width, bottom_diff, bottom_rois);

        err = hipGetLastError();
        if(hipSuccess != err) {
            fprintf( stderr, "cudaCheckError() failed : %s\n", hipGetErrorString( err ) );
            exit( -1 );
        }

        return 1;
    }


#ifdef __cplusplus
}
#endif
